#include "cudamemory.h"
#include "cudaUtility.h"

bool cudaAllocMapped( void** cpuPtr, void** gpuPtr, size_t size )
{
	if( !cpuPtr || !gpuPtr || size == 0 )
		return false;

        if( CUDA_FAILED(hipHostAlloc(cpuPtr, size, hipHostMallocMapped)) )
		return false;

	if( CUDA_FAILED(hipHostGetDevicePointer(gpuPtr, *cpuPtr, 0)) )
		return false;

	memset(*cpuPtr, 0, size);
	printf("[cuda]  cudaAllocMapped %zu bytes, CPU %p GPU %p\n", size, *cpuPtr, *gpuPtr);
	return true;
}
